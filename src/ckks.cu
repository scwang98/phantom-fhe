#include "hip/hip_runtime.h"
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include "ckks.h"
#include "fft.h"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

__global__ void bit_reverse_kernel(hipDoubleComplex *dst, hipDoubleComplex *src, uint64_t in_size,
                                   uint32_t log_n) {
    for (uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < in_size; tid += blockDim.x * gridDim.x) {
        dst[reverse_bits_uint32(tid, log_n)] = src[tid];
    }
}

PhantomCKKSEncoder::PhantomCKKSEncoder(const PhantomContext &context) {
    const auto &s = hipStreamPerThread;

    auto &context_data = context.get_context_data(first_chain_index_);
    auto &parms = context_data.parms();
    std::size_t coeff_count = parms.poly_modulus_degree();

    if (parms.scheme() != scheme_type::ckks) {
        throw std::invalid_argument("unsupported scheme");
    }

    slots_ = coeff_count >> 1;
    uint32_t m = coeff_count << 1;
    uint32_t slots_half = slots_ >> 1;

    gpu_ckks_msg_vec_ = std::make_unique<DCKKSEncoderInfo>(coeff_count, s);

    // We need m powers of the primitive 2n-th root, m = 2n
    root_powers_.reserve(m);
    rotation_group_.reserve(slots_half);

    uint32_t gen = 5;
    uint32_t pos = 1; // Position in normal bit order
    for (size_t i = 0; i < slots_half; i++) {
        // Set the bit-reversed locations
        rotation_group_[i] = pos;

        // Next primitive root
        pos *= gen; // 5^i mod m
        pos &= (m - 1);
    }

    // Powers of the primitive 2n-th root have 4-fold symmetry
    if (m >= 8) {
        complex_roots_ = std::make_unique<util::ComplexRoots>(util::ComplexRoots(static_cast<size_t>(m)));
        for (size_t i = 0; i < m; i++) {
            root_powers_[i] = complex_roots_->get_root(i);
        }
    } else if (m == 4) {
        root_powers_[0] = {1, 0};
        root_powers_[1] = {0, 1};
        root_powers_[2] = {-1, 0};
        root_powers_[3] = {0, -1};
    }

    hipMemcpyAsync(gpu_ckks_msg_vec_->twiddle(), root_powers_.data(), m * sizeof(hipDoubleComplex),
                    hipMemcpyHostToDevice, s);
    hipMemcpyAsync(gpu_ckks_msg_vec_->mul_group(), rotation_group_.data(), slots_half * sizeof(uint32_t),
                    hipMemcpyHostToDevice, s);
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext &context, const std::vector<hipDoubleComplex> &values,
                                         size_t chain_index, double scale,
                                         PhantomPlaintext &destination, const hipStream_t &stream) {
    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    size_t values_size = values.size();

    if (values.empty()) {
        throw std::invalid_argument("Input vector is empty");
    } else if (values_size > slots_) {
        throw std::invalid_argument("Input vector exceeds max slots");
    }

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    auto temp = make_cuda_auto_ptr<hipDoubleComplex>(values_size, stream);
    hipMemcpyAsync(temp.get(), values.data(), values_size * sizeof(hipDoubleComplex), hipMemcpyHostToDevice,
                    stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    size_t gridDimGlb = std::ceil((float) values_size / (float) blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
            gpu_ckks_msg_vec_->in(), temp.get(), values_size, log_slot_count);

    double fix = scale / static_cast<double>(slots_);

    special_fft_backward(*gpu_ckks_msg_vec_, log_slot_count, fix, stream);

    // TODO: boundary check on GPU
    vector<hipDoubleComplex> temp2(slots_);
    hipMemcpyAsync(temp2.data(), gpu_ckks_msg_vec_->in(), slots_ * sizeof(hipDoubleComplex),
                    hipMemcpyDeviceToHost, stream);
    // explicit stream synchronize to avoid error
    hipStreamSynchronize(stream);

    double max_coeff = 0;
    for (std::size_t i = 0; i < slots_; i++) {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < slots_; i++) {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count()) {
        throw std::invalid_argument("encoded values are too large");
    }

    rns_tool.base_Ql().decompose_array(destination.data(), gpu_ckks_msg_vec_->in(), coeff_count, max_coeff_bit_count,
                                       stream);

    nwt_2d_radix8_forward_inplace(destination.data(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext &context, double value, double scale, size_t chain_index,
                                         PhantomPlaintext &destination, hipStream_t const &stream) const {
    auto &context_data = context.get_context_data(chain_index);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t coeff_count = parms.poly_modulus_degree();

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    value *= scale;

    int coeff_bit_count = static_cast<int>(log2(fabs(value))) + 2;
    if (coeff_bit_count >= context_data.total_coeff_modulus_bit_count())
    {
        throw invalid_argument("encoded value is too large");
    }

    double coeffd = round(value);
    bool is_negative = signbit(coeffd);

    uint64_t coeffu = static_cast<uint64_t>(fabs(coeffd));

    if (is_negative)
    {
        for (size_t j = 0; j < coeff_modulus_size; j++)
        {
            thrust::fill_n(thrust::cuda::par.on(stream),
                           thrust::device_ptr<uint64_t>(destination.data()),
                           coeff_count,
                           negate_uint_mod(barrett_reduce_64(coeffu, coeff_modulus[j]), coeff_modulus[j])
            );
        }
    }
    else
    {
        for (size_t j = 0; j < coeff_modulus_size; j++)
        {
            thrust::fill_n(thrust::cuda::par.on(stream),
                           thrust::device_ptr<uint64_t>(destination.data()),
                           coeff_count,
                           barrett_reduce_64(coeffu, coeff_modulus[j])
            );
        }
    }

    destination.chain_index_ = chain_index;
    destination.scale_ = scale;
}

void PhantomCKKSEncoder::decode_internal(const PhantomContext &context, const PhantomPlaintext &plain,
                                         std::vector<hipDoubleComplex> &destination, const hipStream_t &stream) {
    auto &context_data = context.get_context_data(plain.chain_index_);
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    auto &rns_tool = context_data.gpu_rns_tool();
    const size_t coeff_modulus_size = coeff_modulus.size();
    const size_t coeff_count = parms.poly_modulus_degree();
    size_t log_slot_count = arith::get_power_of_two(slots_);
    const size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;

    if (plain.scale() <= 0 ||
        (static_cast<int>(log2(plain.scale())) >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    auto upper_half_threshold = context_data.upper_half_threshold();
    int logn = arith::get_power_of_two(coeff_count);
    auto gpu_upper_half_threshold = make_cuda_auto_ptr<uint64_t>(upper_half_threshold.size(), stream);
    hipMemcpyAsync(gpu_upper_half_threshold.get(), upper_half_threshold.data(),
                    upper_half_threshold.size() * sizeof(uint64_t), hipMemcpyHostToDevice, stream);

    hipMemsetAsync(gpu_ckks_msg_vec_->in(), 0, slots_ * sizeof(hipDoubleComplex), stream);

    // Quick sanity check
    if ((logn < 0) || (coeff_count < POLY_MOD_DEGREE_MIN) || (coeff_count > POLY_MOD_DEGREE_MAX)) {
        throw std::logic_error("invalid parameters");
    }

    double inv_scale = double(1.0) / plain.scale();
    // Create mutable copy of input
    auto plain_copy = make_cuda_auto_ptr<uint64_t>(rns_poly_uint64_count, stream);
    hipMemcpyAsync(plain_copy.get(), plain.data(), rns_poly_uint64_count * sizeof(uint64_t), hipMemcpyDeviceToDevice,
                    stream);

    nwt_2d_radix8_backward_inplace(plain_copy.get(), context.gpu_rns_tables(), coeff_modulus_size, 0, stream);

    // CRT-compose the polynomial
    rns_tool.base_Ql().compose_array(gpu_ckks_msg_vec().in(), plain_copy.get(), gpu_upper_half_threshold.get(),
                                     inv_scale, coeff_count, stream);

    special_fft_forward(*gpu_ckks_msg_vec_, log_slot_count, stream);

    auto out = make_cuda_auto_ptr<hipDoubleComplex>(slots_, stream);
    size_t gridDimGlb = std::ceil((float) slots_ / (float) blockDimGlb.x);
    bit_reverse_kernel<<<gridDimGlb, blockDimGlb, 0, stream>>>(
            out.get(), gpu_ckks_msg_vec_->in(), slots_, log_slot_count);

    destination.resize(slots_);
    hipMemcpyAsync(destination.data(), out.get(), slots_ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);

    // explicit synchronization in case user wants to use the result immediately
    hipStreamSynchronize(stream);
}
